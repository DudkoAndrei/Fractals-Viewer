#include "cuda_error_handler.cuh"

#include <iostream>

void HandleCudaError(hipError_t error) {
  if (error != hipSuccess) {
    std::cerr << hipGetErrorString(error) << "\n";
  }
}
