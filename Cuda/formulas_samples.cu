#include "hip/hip_runtime.h"
#include "complex.cuh"
#include "expression_calculator.cuh"
#include "formulas_samples.cuh"

__global__ void GenerateBWPoint(
    uint64_t* result,
    ImageSettings* settings,
    Calculator<double>* calc) {
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= settings->width * settings->height) {
    return;
  }

  uint64_t y = index / settings->width;
  uint64_t x = index % settings->width;

  Complex<double> c
      ((x - settings->width / 2.0 - settings->offset_x) / settings->scale_x,
       (y - settings->height / 2.0 - settings->offset_y) / settings->scale_y);
  Complex<double> z;

  uint64_t iteration = 0;
  while (iteration < 1000 && z.Abs() < (2 << 8)) {
    z = calc->Calculate(z, c);
    ++iteration;
  }

  if (iteration < 1000) {
    result[index] = iteration % 16 + 1;
  } else {
    result[index] = 0;
  }
}

void CudaBWFractal(
    Array<uint64_t>* data,
    const ImageSettings& settings,
    const std::vector<Token>& expression) {
  uint64_t block_size = 256;
  uint64_t grid_size =
      (settings.width * settings.height + block_size - 1) / block_size;

  ImageSettings* d_settings;  // settings copy, stored in device memory
  hipMalloc(&d_settings, sizeof(ImageSettings));
  hipMemcpy(d_settings,
             &settings,
             sizeof(ImageSettings),
             hipMemcpyHostToDevice);

  Token* d_expression;  // expression copy, stored in device memory
  hipMalloc(&d_expression, sizeof(Token) * expression.size());
  hipMemcpy(d_expression,
             expression.data(),
             sizeof(Token) * expression.size(),
             hipMemcpyHostToDevice);

  Calculator<double> calc(d_expression, expression.size());
  Calculator<double>* d_calc;  // calculator copy, stored in device memory
  hipMalloc(&d_calc, sizeof(Calculator<double>) );
  hipMemcpy(d_calc,
             &calc,
             sizeof(Calculator<double>),
             hipMemcpyHostToDevice);

  GenerateBWPoint<<<grid_size, block_size>>>(data->Data(),
                                             d_settings,
                                             d_calc);

  hipDeviceSynchronize();

  hipFree(d_settings);
}
